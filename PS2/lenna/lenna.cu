#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "lodepng.h"

#define subpixel_t unsigned char
#define VALUES_PER_PIXEL 3 // no alpha channel when using decode24, just RGB values

__global__
void kernel(subpixel_t* image) {
    /*
       Each thread works on (3 values == 1 pixel)
     */
    unsigned int x = (blockIdx.x * blockDim.x + threadIdx.x);
    unsigned int y = (blockIdx.y * blockDim.y + threadIdx.y);
    unsigned int w = gridDim.x * blockDim.x;
    unsigned int i = VALUES_PER_PIXEL * (w * y + x);

    for (int o = 0; o < VALUES_PER_PIXEL; o++)  {
        image[i+o] = ~image[i+o];
    }
}

int main( int argc, char ** argv){
    size_t pngsize;
    subpixel_t *png;
    const char * filename = "lenna512x512_inv.png";

    lodepng_load_file(&png, &pngsize, filename);

    subpixel_t *image;
    unsigned int width, height;
    unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

    if (error != 0){
        std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
    }

    unsigned int size = sizeof(subpixel_t) * width * height * VALUES_PER_PIXEL;

    dim3 blockDim(16, 16);
    dim3 gridDim(width/blockDim.x, height/blockDim.y);

    subpixel_t* device_a;
    hipMalloc ((void **)&device_a, size);

    float dt_0_1, dt_2_3;
    hipEvent_t t0, t1, t2, t3;
    hipEventCreate(&t0);
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);

    hipEventRecord(t0, 0); 
    hipMemcpy(device_a, image, size, hipMemcpyHostToDevice);
    hipEventRecord(t1, 0); 
    hipEventSynchronize(t1);
    hipEventElapsedTime(&dt_0_1, t0, t1);

    kernel<<<gridDim, blockDim>>>(device_a);

    hipEventRecord(t2, 0); 
    hipMemcpy(image, device_a, size, hipMemcpyDeviceToHost);
    hipEventRecord(t3, 0); 
    hipEventSynchronize(t3);
    hipEventElapsedTime(&dt_2_3, t2, t3);

    printf("total transfer time: %f ms\n", dt_0_1 + dt_2_3);

    lodepng_encode24_file("lenna512x512_orig.png", image, width, height);

    free(image);
    hipFree(device_a);

    return 0;
}

