#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "lodepng.h"

#define subpixel_t unsigned char
#define VALUES_PER_PIXEL 3 // no alpha channel when using decode24, just RGB values

__global__
void kernel(subpixel_t* image) {
    /*
        Each thread works on (3 values == 1 pixel)
    */
    unsigned int x = (blockIdx.x * blockDim.x + threadIdx.x);
    unsigned int y = (blockIdx.y * blockDim.y + threadIdx.y);
    unsigned int w = gridDim.x * blockDim.x;
    unsigned int i = VALUES_PER_PIXEL * (w * y + x);

    for (int o = 0; o < VALUES_PER_PIXEL; o++)  {
        image[i+o] = ~image[i+o];
    }
}

int main( int argc, char ** argv){

    size_t pngsize;
    subpixel_t *png;
    const char * filename = "lenna512x512_inv.png";

    /* Read in the image */
    lodepng_load_file(&png, &pngsize, filename);

    /* Decode it into a RGB 8-bit per channel vector */
    subpixel_t *image;
    unsigned int width, height;
    unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

    /* Check if read and decode of .png went well */
    if(error != 0){
        std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
    }

    // Do work

    unsigned int size = sizeof(subpixel_t) * width * height * VALUES_PER_PIXEL;

    dim3 blockDim(16, 16);
    dim3 gridDim(width/blockDim.x, height/blockDim.y);

    subpixel_t* device_a;
    hipMalloc ((void **)&device_a, size);

    hipMemcpy(device_a, image, size, hipMemcpyHostToDevice);
    kernel<<<gridDim, blockDim>>>(device_a);
    hipMemcpy(image, device_a, size, hipMemcpyDeviceToHost);

    /* Save the result to a new .png file */
    lodepng_encode24_file("lenna512x512_orig.png", image, width, height);

    free(image);
    hipFree(device_a);

    return 0;
}

